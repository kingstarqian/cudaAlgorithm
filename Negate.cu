#include "hip/hip_runtime.h"
#include "Negate.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Common.h"

__global__ void negateAlgorithm(unsigned short * deviceBuffer)
{
	int i = threadIdx.x;
	deviceBuffer[i] = MAX_PIXEL_VALUE - deviceBuffer[i];
}

Negate::Negate()
{
}


Negate::~Negate()
{
}

void Negate::fire(unsigned short * deviceBuffer)
{
	CALCULATE_CUDA_ELAPSED_START

	for (int i = 0; i < IMAGE_HEIGHT; ++i)
	{
		negateAlgorithm <<<1, IMAGE_WIDTH >>>(deviceBuffer + IMAGE_WIDTH * i);
	}

	dim3 dim(IMAGE_WIDTH, IMAGE_HEIGHT);

	CALCULATE_CUDA_ELAPSED_STOP("Negate")
	
	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Negate::fire failed!");
	}
}
