#include "hip/hip_runtime.h"
#include "Negate.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Common.h"

__global__ void negateAlgorithm(unsigned short * deviceBuffer)
{
	int i = threadIdx.x;
	deviceBuffer[i] = MAX_PIXEL_VALUE - deviceBuffer[i];
}

Negate::Negate()
{
}


Negate::~Negate()
{
}

void Negate::fire(unsigned short * deviceBuffer)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int i = 0; i < IMAGE_HEIGHT; ++i)
	{
		negateAlgorithm << <1, IMAGE_WIDTH >> >(deviceBuffer + IMAGE_WIDTH * i);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float tm;
	hipEventElapsedTime(&tm, start, stop);
	printf("Negate GPU Elapsed time:%.6f ms.\n", tm);
	
	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Negate::fire failed!");
	}
}
