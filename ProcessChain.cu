#include "ProcessChain.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "Common.h"
#include "Negate.h"
#include "RawImageData.h"

using namespace std;

ProcessChain::ProcessChain(): deviceBuffer(NULL),negate(NULL)
{
}

ProcessChain::~ProcessChain()
{
}

bool ProcessChain::initializeCuda()
{
	if (initializeDevice())
	{
		mallocMemory();
		initializeChain();
	}
	else
	{
		return false;
	}

	return true;
}


bool ProcessChain::initializeDevice()
{
	hipError_t cudaStatus;

	int count = 0;
	cudaStatus = hipGetDeviceCount(&count);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?");
		return false;
	}
	if (count != 1)
	{
		cout << "Cuda device count is not one. count = " << count << endl;
		return false;
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!");
		return false;
	}

	//Get Device Properties
	hipDeviceProp_t prop;
	cudaStatus = hipGetDeviceProperties(&prop, 0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Get CUDA devices failed!");
		return false;
	}

	cout << "MultiProcessCount: " << prop.multiProcessorCount << endl;
	cout << "MaxThreadsPerBlock: " << prop.maxThreadsPerBlock << endl;
	cout << "MaxThreadsPerMultiProcessor" << prop.maxThreadsPerMultiProcessor << endl;

	return true;
}

bool ProcessChain::mallocMemory()
{
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&deviceBuffer, IMAGE_SIZE * sizeof(unsigned short));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Malloc device memory failed!");
		return false;
	}

	return true;
}


void ProcessChain::uninitializeCuda()
{
	freeMemory();
}

void ProcessChain::process(RawImageData* image)
{
	hipMemcpy(deviceBuffer, image->getBuffer(), IMAGE_SIZE * sizeof(unsigned short), hipMemcpyHostToDevice);
	negate->fire(deviceBuffer);
	hipMemcpy(image->getBuffer(), deviceBuffer, IMAGE_SIZE * sizeof(unsigned short), hipMemcpyDeviceToHost);
}

void ProcessChain::freeMemory()
{

}

void ProcessChain::initializeChain()
{
	negate = new Negate();
}
